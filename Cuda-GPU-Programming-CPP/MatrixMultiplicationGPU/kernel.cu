#include "hip/hip_runtime.h"
#ifndef Kernel_CU
#define Kernel_CU

#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
using namespace std;

__global__ void matrixMultiplicationKernel(float *A, float *B, float *C, int N)
{
	int Col = blockIdx.y*blockDim.y+ threadIdx.y;
	int Row = blockIdx.x*blockDim.x+ threadIdx.x;

	float tmpSum = 0;

	if(Row<N&&Col<N)
	{
		for(int i=0;i<N;i++)
		{
			tmpSum += A[Row*N+i] * B[i*N+Col];
		}
	}
	C[Row*N+Col] = tmpSum;
}

void matrixMultiplication(float *A, float *B, float *C, int N)
{
	dim3 threadsPerBlock(N,N);
	dim3 blocksPerGrid(1,1);
	if(N*N>1024)
	{
		threadsPerBlock.x = 64;
		threadsPerBlock.y = 64;
		blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
	}

	matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
}

#endif
